#include <iostream>
#include <hip/hip_runtime.h>

inline const char *_ConvertSMVer2ArchName(int major, int minor) {
    // Defines for GPU Architecture types (using the SM version to determine
    // the GPU Arch name)
    typedef struct {
        int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
        // and m = SM minor version
        const char *name;
    } sSMtoArchName;

    sSMtoArchName nGpuArchNameSM[] = {
            {0x30, "Kepler"},
            {0x32, "Kepler"},
            {0x35, "Kepler"},
            {0x37, "Kepler"},
            {0x50, "Maxwell"},
            {0x52, "Maxwell"},
            {0x53, "Maxwell"},
            {0x60, "Pascal"},
            {0x61, "Pascal"},
            {0x62, "Pascal"},
            {0x70, "Volta"},
            {0x72, "Xavier"},
            {0x75, "Turing"},
            {0x80, "Ampere"},
            {0x86, "Ampere"},
            {-1,   "Graphics Device"}};

    int index = 0;

    while (nGpuArchNameSM[index].SM != -1) {
        if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchNameSM[index].name;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one
    // to run properly
    printf(
            "MapSMtoArchName for SM %d.%d is undefined."
            "  Default to use %s\n",
            major, minor, nGpuArchNameSM[index - 1].name);
    return nGpuArchNameSM[index - 1].name;
}
// end of GPU Architecture definitions

int main(void) {
    hipDeviceProp_t devProp{};
    hipGetDeviceProperties(&devProp, 0);
    std::cout << "使用GPU device " << 0 << ": " << devProp.name << std::endl;
    std::cout << "GPU Architecture: " << _ConvertSMVer2ArchName(devProp.major, devProp.minor) << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "SM算力：" << devProp.major << "." << devProp.minor << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
    return 0;
}